
#include <hip/hip_runtime.h>
// Device code
extern "C" __global__ void l2distance_f32(
        double *res, int n, int vecsz, bool sq,
        const float *A, const float *B) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int loop = vecsz / sizeof(float);
    if (i < n) {
        res[i] = 0;
        for (int j = 0; j < loop; j++) {
            float diff = A[i * loop + j] - B[i * loop + j];
            res[i] += diff * diff;
        }
        if (!sq) {
            res[i] = sqrt(res[i]);
        }
    }
}

extern "C" __global__ void l2distance_f32_const(
        double *res, int n, int vecsz, bool sq,
        const float *A, const float *B) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int loop = vecsz / sizeof(float);
    if (i < n) {
        res[i] = 0;
        for (int j = 0; j < loop; j++) {
            float diff = A[i * loop + j] - B[j];
            res[i] += diff * diff;
        }
        if (!sq) {
            res[i] = sqrt(res[i]);
        }
    }
}

extern "C" __global__ void l2distance_f64(
        double *res, int n, int vecsz, bool sq,
        const double *A, const double *B) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int loop = vecsz / sizeof(double);
    if (i < n) {
        res[i] = 0;
        for (int j = 0; j < loop; j++) {
            double diff = A[i * loop + j] - B[i * loop + j];
            res[i] += diff * diff;
        }
        if (!sq) {
            res[i] = sqrt(res[i]);
        } 
    }
}

extern "C" __global__ void l2distance_f64_const(
        double *res, int n, int vecsz, bool sq,
        const float *A, const float *B) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int loop = vecsz / sizeof(double);
    res[i] = 0;
    if (i < n) {
        for (int j = 0; j < loop; j++) {
            double diff = A[i * loop + j] - B[j];
            res[i] += diff * diff;
        }
        if (!sq) {
            res[i] = sqrt(res[i]);
        }
    }
}
